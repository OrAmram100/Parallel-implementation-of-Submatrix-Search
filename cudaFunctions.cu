#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <string.h>
#include "cudaHeader.h"

/**** GPU ****/

/**** calculate the diff between pic cell and obj cell ****/
__device__ double fabsFunc(int p, int o)
{
	if(p==0)
		return 0;
	double diff = fabs((double)(p-o)/p);
		return diff;
}

/**** this function calculate the res from matching(row,col) ****/
__device__ double calcRes(int objectSize,int pictureSize,int* picture, int* object, int row,int col,double matchingValue) 
{
	double res = 0;
	for (int i = 0; i < objectSize; i++, row++) {
		int indexCol = col;
		for (int j = 0; j < objectSize; j++, indexCol++) {
			res += fabsFunc(picture[row*pictureSize +indexCol],object[i*objectSize + j]);
			
		}
	}
	return res;
}
/**** this function calculate the matching sum and put it into 'sumArray'****/

__global__ void calcSum( int pictureSize,int objectSize,int* picture,int* object, double* SumArray,double matchingValue) {
	double matching = 0;
	for(int i = threadIdx.x ; i < pictureSize; i += NUM_THREADS_PER_BLOCK) {
		for(int j = threadIdx.y ; j < pictureSize; j += NUM_THREADS_PER_BLOCK) {				
			matching = 0;
			matching = calcRes(objectSize,pictureSize,picture,object,i,j,matchingValue);
				
			SumArray[i*pictureSize + j] = matching;
			
		}
	}
}


void computeOnGPU(obj picture,obj* objects,int numberOfObjects,double matchingValue,int* numOfMatching,idealMatching* idealMatch)
{
	int flag = 0;
	idealMatch->objectID = -1;	
    int *d_Picture;
	
    // Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;
  
    // Allocate memory on GPU to copy the data from the host
    	    	
    err = hipMalloc((void **)&d_Picture, (picture.size * picture.size) * sizeof(int));
    if (err != hipSuccess) {

        fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	
    // Copy data from host to the GPU memory  	
    err = hipMemcpy(d_Picture, picture.members, (picture.size * picture.size) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
	fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
	exit(EXIT_FAILURE);
    }
    /**** pass with each object whether it is in the picture  ****/
    for(int i = 0; i < numberOfObjects; i++) {
		int *d_Object;
		/****h_SumArray will show all the potential sums that the GPU has calculated ****/
		double *d_SumArray, *h_SumArray;
		int pictureTotalSize = picture.size*picture.size;
		int objectTotalSize = objects[i].size*objects[i].size;
    				
		// Allocate memory on GPU to copy the data from the host
    	err = hipMalloc((void **)&d_Object, objectTotalSize * sizeof(int));
    	if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
    	// Allocate memory on GPU to copy the data from the host
    	err = hipMalloc((void **)&d_SumArray, pictureTotalSize* sizeof(double));
		if (err != hipSuccess) {

			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

    	// Copy data from host to the GPU memory  	
		err = hipMemcpy(d_Object, objects[i].members, objectTotalSize * sizeof(int), hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to copy data from host to device - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}

		// Launch the Kernel
		/**** if the size of the picture is smaller than 32 then we will run a single block with size pictureSize X pictureSize threads ****/
		if(picture.size <= NUM_THREADS_PER_BLOCK) {
			dim3 dimBlock(picture.size, picture.size);
			calcSum<<<1, dimBlock>>>(picture.size, objects[i].size, d_Picture,d_Object, d_SumArray,matchingValue);
			err = hipGetLastError();
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch calcSum kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}

			}
			/**** if the size of the picture is bigger than 32 then we will send max block with size 32 X 32 threads ****/
		else {
			dim3 dimBlock(NUM_THREADS_PER_BLOCK, NUM_THREADS_PER_BLOCK);
			calcSum<<<1, dimBlock>>>(picture.size, objects[i].size, d_Picture,d_Object, d_SumArray,matchingValue);
			err = hipGetLastError();
			if (err != hipSuccess)
			{
				fprintf(stderr, "Failed to launch calcSum kernel (error code %s)!\n", hipGetErrorString(err));
				exit(EXIT_FAILURE);
			}
			}
				
		h_SumArray = (double*)malloc(pictureTotalSize* sizeof(double));
		if (h_SumArray == NULL) {
			printf("Problem to allocate memory\n");
			exit(0);
			}
		// Allocate memory on GPU to copy the data from the GPU to CPU
			err = hipMemcpy(h_SumArray, d_SumArray, pictureTotalSize * sizeof(double), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf(stderr, "Failed to allocate device memory - %s\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
				}
				
		// check for all index in h_summarray if sum <= matching value					
		for(int x = 0; x < picture.size && !flag; x++) 
			for(int y = 0; y <picture.size && !flag; y++) 
				if(h_SumArray[x*picture.size + y] <= matchingValue) {
					flag=1;								
					*numOfMatching+=1;
					idealMatch->i = x;
					idealMatch->j = y;
					idealMatch->pictureID = picture.id;
					idealMatch->objectID = objects[i].id;
					}
				
			
				
		// Free allocated memory on GPU
		err = hipFree(d_SumArray);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to free device d_SumArray (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
		 // Free allocated memory on GPU
		err = hipFree(d_Object);
		if (err != hipSuccess)
		{
			fprintf(stderr, "Failed to free device d_Object (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
		}
							
		free(h_SumArray);			
		 if(flag ==1)
			   break;
	}

    // Free allocated memory on GPU
    err = hipFree(d_Picture);
    if (err != hipSuccess)
	{
			fprintf(stderr, "Failed to free device d_SumArray (error code %s)!\n", hipGetErrorString(err));
			exit(EXIT_FAILURE);
	}
}
